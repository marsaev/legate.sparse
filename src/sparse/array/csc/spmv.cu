/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/array/csc/spmv.h"
#include "sparse/array/csc/spmv_template.inl"
#include "sparse/util/cusparse_utils.h"
#include "sparse/util/dispatch.h"

namespace sparse {

template <>
struct CSCSpMVColSplitImpl<VariantKind::GPU> {
  template <LegateTypeCode INDEX_CODE, LegateTypeCode VAL_CODE>
  void operator()(CSCSpMVColSplitArgs& args) const
  {
    using INDEX_TY = legate_type_of<INDEX_CODE>;
    using VAL_TY   = legate_type_of<VAL_CODE>;

    auto& y      = args.y;
    auto& A_pos  = args.A_pos;
    auto& A_crd  = args.A_crd;
    auto& A_vals = args.A_vals;
    auto& x      = args.x;

    // Break out early if the iteration space partition is empty.
    if (x.domain().empty()) return;

    // Get context sensitive objects.
    auto handle = get_cusparse();
    auto stream = get_cached_stream();
    CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    // Construct the CUSPARSE objects from individual regions.
    auto cusparse_y = makeCuSparseDenseVec<VAL_TY>(y);
    auto cusparse_x = makeCuSparseDenseVec<VAL_TY>(x);
    auto cusparse_A =
      makeCuSparseCSC<INDEX_TY, VAL_TY>(A_pos, A_crd, A_vals, y.domain().get_volume() /* rows */);

    // Make the CUSPARSE calls.
    VAL_TY alpha   = 1.0;
    VAL_TY beta    = 0.0;
    size_t bufSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           cusparse_A,
                                           cusparse_x,
                                           &beta,
                                           cusparse_y,
                                           cusparseDataType<VAL_TY>(),
#if (CUSPARSE_VER_MAJOR < 11 || CUSPARSE_VER_MINOR < 2)
                                           HIPSPARSE_MV_ALG_DEFAULT,
#else
                                           HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                                           &bufSize));
    // Allocate a buffer if we need to.
    void* workspacePtr = nullptr;
    if (bufSize > 0) {
      DeferredBuffer<char, 1> buf({0, bufSize - 1}, Memory::GPU_FB_MEM);
      workspacePtr = buf.ptr(0);
    }
    // Finally do the SpMV.
    CHECK_CUSPARSE(hipsparseSpMV(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                cusparse_A,
                                cusparse_x,
                                &beta,
                                cusparse_y,
                                cusparseDataType<VAL_TY>(),
#if (CUSPARSE_VER_MAJOR < 11 || CUSPARSE_VER_MINOR < 2)
                                HIPSPARSE_MV_ALG_DEFAULT,
#else
                                HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                                workspacePtr));
    // Destroy the created objects.
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_y));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_x));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_A));
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void CSCSpMVColSplit::gpu_variant(legate::TaskContext& context)
{
  csc_spmv_col_split_template<VariantKind::GPU>(context);
}

}  // namespace sparse
