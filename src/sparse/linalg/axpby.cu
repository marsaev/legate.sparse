#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/linalg/axpby.h"
#include "sparse/linalg/axpby_template.inl"
#include "sparse/util/cuda_help.h"

namespace sparse {

using namespace Legion;
using namespace legate;

template <typename VAL_TY, bool IS_ALPHA>
__global__ void axpby_kernel(size_t elems,
                             coord_t offset,
                             AccessorRW<VAL_TY, 1> y,
                             AccessorRO<VAL_TY, 1> x,
                             AccessorRO<VAL_TY, 1> alphabeta)
{
  const auto idx = global_tid_1d();
  if (idx >= elems) return;
  auto i = idx + offset;
  if (IS_ALPHA) {
    y[i] = alphabeta[0] * x[i] + y[i];
  } else {
    y[i] = x[i] + alphabeta[0] * y[i];
  }
}

template <LegateTypeCode VAL_CODE, bool IS_ALPHA>
struct AXPBYImplBody<VariantKind::GPU, VAL_CODE, IS_ALPHA> {
  using VAL_TY = legate_type_of<VAL_CODE>;

  void operator()(const AccessorRW<VAL_TY, 1>& y,
                  const AccessorRO<VAL_TY, 1>& x,
                  const AccessorRO<VAL_TY, 1>& alphabeta,
                  const Rect<1>& rect)
  {
    auto elems  = rect.volume();
    auto blocks = get_num_blocks_1d(elems);
    auto stream = get_cached_stream();
    axpby_kernel<VAL_TY, IS_ALPHA>
      <<<blocks, THREADS_PER_BLOCK, 0, stream>>>(elems, rect.lo[0], y, x, alpha);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void AXPBY::gpu_variant(TaskContext& context)
{
  axpby_template<VariantKind::GPU>(context);
}

}  // namespace sparse
