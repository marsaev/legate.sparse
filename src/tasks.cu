#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse.h"
#include "tasks.h"
#include "cuda_help.h"
#include "pitches.h"
#include "distal_cuda_utils.h"
#include "sparse/util/cusparse_utils.h"

#include "thrust_allocator.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

namespace sparse {

using namespace legate;
using namespace Legion;

const hipDataType cuda_val_ty   = HIP_R_64F;
const hipsparseIndexType_t index_ty = HIPSPARSE_INDEX_64I;

// We also include an overload that does not specify the values.
hipsparseSpMatDescr_t makeCuSparseCSR(Store& pos, Store& crd, size_t cols)
{
  hipsparseSpMatDescr_t matDescr;
  auto stream = get_cached_stream();

  auto pos_domain = pos.domain();
  auto crd_domain = crd.domain();

  auto pos_acc = pos.read_accessor<Rect<1>, 1>();
  size_t rows  = pos_domain.get_volume();
  DeferredBuffer<int64_t, 1> indptr({0, rows}, Memory::GPU_FB_MEM);
  auto blocks = get_num_blocks_1d(rows);
  convertGlobalPosToLocalIndPtr<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
    rows, pos_acc.ptr(pos_domain.lo()), indptr.ptr(0));

  CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr,
                                   rows,
                                   cols,
                                   crd_domain.get_volume(), /* nnz */
                                   (void*)indptr.ptr(0),
                                   crd_domain.empty() ? nullptr : getPtrFromStore<coord_ty, 1>(crd),
                                   nullptr,
                                   index_ty,
                                   index_ty,
                                   index_base,
                                   cuda_val_ty));

  return matDescr;
}

__global__ void offset_coordinates_to_global(size_t elems, coord_ty offset, coord_ty* coords)
{
  const auto idx = global_tid_1d();
  if (idx >= elems) return;
  coords[idx] += offset;
}

void SpGEMMCSRxCSRxCSCLocalTiles::gpu_variant(legate::TaskContext& ctx)
{
  auto& A_pos  = ctx.outputs()[0];
  auto& A_crd  = ctx.outputs()[1];
  auto& A_vals = ctx.outputs()[2];

  auto& B_pos  = ctx.inputs()[0];
  auto& B_crd  = ctx.inputs()[1];
  auto& B_vals = ctx.inputs()[2];
  auto& C_pos  = ctx.inputs()[3];
  auto& C_crd  = ctx.inputs()[4];
  auto& C_vals = ctx.inputs()[5];

  // TODO (rohany): Make sure to package these scalars in to the task.
  int64_t C_rows = ctx.scalars()[0].value<int64_t>();

  // Get context sensitive objects.
  auto handle = get_cusparse();
  auto stream = get_cached_stream();
  CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

  // Remove the transformations on the B_pos and C_pos stores.
  B_pos.remove_transform();
  C_pos.remove_transform();

  // Par for the course, neither the conversion routines or SpGEMM computations
  // allow for 64-bit integer coordinates. So we have to convert everything
  // into 64-bit integers. Next, the SpGEMM algorithm only supports CSR matrices!
  // So we have to convert the C CSC matrix into a local tile of CSR before we can
  // even do the multiply.
  auto B_rows = B_pos.domain().get_volume();
  auto C_cols = C_pos.domain().get_volume();

  // Start doing the casts.
  DeferredBuffer<int32_t, 1> B_indptr({0, B_rows}, Memory::GPU_FB_MEM);
  DeferredBuffer<int32_t, 1> C_indptr({0, C_cols}, Memory::GPU_FB_MEM);
  {
    auto blocks = get_num_blocks_1d(B_rows);
    convertGlobalPosToLocalIndPtr<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      B_rows, B_pos.read_accessor<Rect<1>, 1>().ptr(B_pos.domain().lo()), B_indptr.ptr(0));
  }
  {
    auto blocks = get_num_blocks_1d(C_cols);
    convertGlobalPosToLocalIndPtr<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      C_cols, C_pos.read_accessor<Rect<1>, 1>().ptr(C_pos.domain().lo()), C_indptr.ptr(0));
  }
  DeferredBuffer<int32_t, 1> B_crd_int({0, B_crd.domain().get_volume() - 1}, Memory::GPU_FB_MEM);
  DeferredBuffer<int32_t, 1> C_crd_int({0, C_crd.domain().get_volume() - 1}, Memory::GPU_FB_MEM);
  {
    auto dom    = B_crd.domain();
    auto elems  = dom.get_volume();
    auto blocks = get_num_blocks_1d(elems);
    cast<int, coord_ty><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      elems, B_crd_int.ptr(0), B_crd.read_accessor<coord_ty, 1>().ptr(dom.lo()));
  }
  {
    auto dom    = C_crd.domain();
    auto elems  = dom.get_volume();
    auto blocks = get_num_blocks_1d(elems);
    cast<int, coord_ty><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      elems, C_crd_int.ptr(0), C_crd.read_accessor<coord_ty, 1>().ptr(dom.lo()));
  }
  // Now, we can start the conversion of C from CSC to CSR. The method is called
  // CSR2CSC, so we can use it in the reverse way also by doing CSC2CSR.
  // First, allocate buffers for the resulting C CSR data.
  DeferredBuffer<int32_t, 1> C_CSR_indptr({0, C_rows}, Memory::GPU_FB_MEM);
  DeferredBuffer<int32_t, 1> C_CSR_crd({0, C_crd.domain().get_volume() - 1}, Memory::GPU_FB_MEM);
  DeferredBuffer<val_ty, 1> C_CSR_vals({0, C_crd.domain().get_volume() - 1}, Memory::GPU_FB_MEM);
  size_t convBufSize = 0;
  CHECK_CUSPARSE(hipsparseCsr2cscEx2_bufferSize(handle,
                                               // Because we're treating the CSC matrix as a CSR
                                               // matrix, we have to reverse the columns and rows.
                                               C_cols,
                                               C_rows,
                                               C_crd.domain().get_volume(),
                                               getPtrFromStore<val_ty, 1>(C_vals),
                                               C_indptr.ptr(0),
                                               C_crd_int.ptr(0),
                                               C_CSR_vals.ptr(0),
                                               C_CSR_indptr.ptr(0),
                                               C_CSR_crd.ptr(0),
                                               cuda_val_ty,
                                               HIPSPARSE_ACTION_NUMERIC,
                                               index_base,
                                               HIPSPARSE_CSR2CSC_ALG1,
                                               &convBufSize));
  DeferredBuffer<char*, 1> convBuffer({0, convBufSize - 1}, Memory::GPU_FB_MEM);
  CHECK_CUSPARSE(hipsparseCsr2cscEx2(handle,
                                    // Look above for reasoning about the size reversal.
                                    C_cols,
                                    C_rows,
                                    C_crd.domain().get_volume(),
                                    getPtrFromStore<val_ty, 1>(C_vals),
                                    C_indptr.ptr(0),
                                    C_crd_int.ptr(0),
                                    C_CSR_vals.ptr(0),
                                    C_CSR_indptr.ptr(0),
                                    C_CSR_crd.ptr(0),
                                    cuda_val_ty,
                                    HIPSPARSE_ACTION_NUMERIC,
                                    index_base,
                                    HIPSPARSE_CSR2CSC_ALG1,
                                    convBuffer.ptr(0)));
  // We don't need this instance anymore.
  convBuffer.destroy();

  // Now we can do the SpGEMM. First, create all of the cusparse matrices.
  hipsparseSpMatDescr_t cusparse_A, cusparse_B, cusparse_C;
  CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_B,
                                   B_rows,
                                   C_rows /* cols */,
                                   B_crd.domain().get_volume() /* nnz */,
                                   B_indptr.ptr(0),
                                   B_crd_int.ptr(0),
                                   getPtrFromStore<val_ty, 1>(B_vals),
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   index_base,
                                   cuda_val_ty));
  CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_C,
                                   C_rows,
                                   C_cols,
                                   C_crd.domain().get_volume() /* nnz */,
                                   C_CSR_indptr.ptr(0),
                                   C_CSR_crd.ptr(0),
                                   C_CSR_vals.ptr(0),
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   index_base,
                                   cuda_val_ty));
  CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_A,
                                   B_rows /* rows */,
                                   C_cols /* cols */,
                                   0 /* nnz */,
                                   nullptr,
                                   nullptr,
                                   nullptr,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   index_base,
                                   cuda_val_ty));

  // Allocate the SpGEMM descriptor.
  hipsparseSpGEMMDescr_t descr;
  CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&descr));

  val_ty alpha = 1.0, beta = 0.0;
  size_t bufferSize1 = 0, bufferSize2 = 0;
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               cusparse_B,
                                               cusparse_C,
                                               &beta,
                                               cusparse_A,
                                               cuda_val_ty,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               descr,
                                               &bufferSize1,
                                               nullptr));
  void* buffer1 = nullptr;
  if (bufferSize1 > 0) {
    DeferredBuffer<char, 1> buf({0, bufferSize1 - 1}, Memory::GPU_FB_MEM);
    buffer1 = buf.ptr(0);
  }
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               cusparse_B,
                                               cusparse_C,
                                               &beta,
                                               cusparse_A,
                                               cuda_val_ty,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               descr,
                                               &bufferSize1,
                                               buffer1));
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        cusparse_B,
                                        cusparse_C,
                                        &beta,
                                        cusparse_A,
                                        cuda_val_ty,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        descr,
                                        &bufferSize2,
                                        nullptr));
  void* buffer2 = nullptr;
  if (bufferSize2 > 0) {
    DeferredBuffer<char, 1> buf({0, bufferSize2 - 1}, Memory::GPU_FB_MEM);
    buffer2 = buf.ptr(0);
  }
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        cusparse_B,
                                        cusparse_C,
                                        &beta,
                                        cusparse_A,
                                        cuda_val_ty,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        descr,
                                        &bufferSize2,
                                        buffer2));
  // Allocate buffers for the 32-bit version of the A matrix.
  int64_t A_rows, A_cols, A_nnz;
  CHECK_CUSPARSE(hipsparseSpMatGetSize(cusparse_A, &A_rows, &A_cols, &A_nnz));
  DeferredBuffer<int32_t, 1> A_indptr({0, A_rows}, Memory::GPU_FB_MEM);
  DeferredBuffer<int32_t, 1> A_crd_int({0, A_nnz - 1}, Memory::GPU_FB_MEM);
  auto A_vals_acc = A_vals.create_output_buffer<val_ty, 1>(A_nnz, true /* return_buffer */);
  CHECK_CUSPARSE(
    hipsparseCsrSetPointers(cusparse_A, A_indptr.ptr(0), A_crd_int.ptr(0), A_vals_acc.ptr(0)));
  CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     cusparse_B,
                                     cusparse_C,
                                     &beta,
                                     cusparse_A,
                                     cuda_val_ty,
                                     HIPSPARSE_SPGEMM_DEFAULT,
                                     descr));

  // Convert the A_indptr array into a pos array.
  {
    auto blocks = get_num_blocks_1d(A_rows);
    auto buf    = A_pos.create_output_buffer<Rect<1>, 1>(A_rows, true /* return_buffer */);
    localIndptrToPos<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(A_rows, buf.ptr(0), A_indptr.ptr(0));
  }
  // Cast the A coordinates back into 64 bits.
  {
    auto blocks = get_num_blocks_1d(A_nnz);
    auto buf    = A_crd.create_output_buffer<coord_ty, 1>(A_nnz, true /* return_buffer */);
    cast<coord_ty, int>
      <<<blocks, THREADS_PER_BLOCK, 0, stream>>>(A_nnz, buf.ptr(0), A_crd_int.ptr(0));
    // Finally, we need to offset the resulting coordinates into the global space.
    // cuSPARSE is going to compute a resulting matrix where all the coordinates
    // are zero-based, but we need the coordinates to be global addressable, so we
    // offset them by the partition of the column space that we are in.
    offset_coordinates_to_global<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      A_nnz, C_pos.domain().lo()[0], buf.ptr(0));
  }
  CHECK_CUDA_STREAM(stream);
}

__global__ void calculate_copy_sizes(size_t total_rows,
                                     size_t num_rects,
                                     DeferredBuffer<Rect<1>, 1> rects,
                                     DeferredBuffer<size_t, 1> row_offsets,
                                     AccessorRO<Rect<1>, 1> global_pos_acc)
{
  const auto idx = global_tid_1d();
  if (idx >= total_rows) return;
  size_t elems = 0;
  for (size_t i = 0; i < num_rects; i++) {
    auto rect           = rects[i];
    auto global_pos_idx = rect.lo + idx;
    if (rect.contains(global_pos_idx)) { elems += global_pos_acc[global_pos_idx].volume(); }
  }
  row_offsets[idx] = elems;
}

__global__ void scatter_rows(size_t total_rows,
                             size_t num_rects,
                             DeferredBuffer<Rect<1>, 1> rects,
                             DeferredBuffer<size_t, 1> row_offsets,
                             AccessorRO<Rect<1>, 1> global_pos_acc,
                             AccessorRO<coord_ty, 1> global_crd_acc,
                             AccessorRO<val_ty, 1> global_vals_acc,
                             DeferredBuffer<Rect<1>, 1> pos_acc,
                             DeferredBuffer<coord_ty, 1> crd_acc,
                             DeferredBuffer<val_ty, 1> vals_acc)
{
  const auto idx = global_tid_1d();
  if (idx >= total_rows) return;
  auto offset = row_offsets[idx];
  auto lo     = offset;
  for (size_t i = 0; i < num_rects; i++) {
    auto rect           = rects[i];
    auto global_pos_idx = rect.lo + idx;
    if (rect.contains(global_pos_idx)) {
      for (int64_t pos = global_pos_acc[global_pos_idx].lo;
           pos < global_pos_acc[global_pos_idx].hi + 1;
           pos++) {
        crd_acc[offset]  = global_crd_acc[pos];
        vals_acc[offset] = global_vals_acc[pos];
        offset++;
      }
    }
  }
  auto hi      = offset - 1;
  pos_acc[idx] = {lo, hi};
}

void SpGEMMCSRxCSRxCSCShuffle::gpu_variant(legate::TaskContext& ctx)
{
  auto& global_pos  = ctx.inputs()[0];
  auto& global_crd  = ctx.inputs()[1];
  auto& global_vals = ctx.inputs()[2];

  auto& out_pos  = ctx.outputs()[0];
  auto& out_crd  = ctx.outputs()[1];
  auto& out_vals = ctx.outputs()[2];

  // TODO (rohany): I want a sparse instance here.
  auto global_pos_acc  = global_pos.read_accessor<Rect<1>, 1>();
  auto global_crd_acc  = global_crd.read_accessor<coord_ty, 1>();
  auto global_vals_acc = global_vals.read_accessor<val_ty, 1>();
  auto stream          = get_cached_stream();

  // Collect all rectangles in the global_pos domain.
  std::vector<Rect<1>> rects_cpu;
  size_t total_nnzs = 0;
  size_t total_rows = 0;
  for (RectInDomainIterator<1> itr(global_pos.domain()); itr(); itr++) {
    rects_cpu.push_back(*itr);
    total_rows = std::max(itr->volume(), total_rows);
    if (itr->empty()) continue;
    Rect<1> lo, hi;
    hipMemcpy(&lo, global_pos_acc.ptr(itr->lo), sizeof(Rect<1>), hipMemcpyDeviceToHost);
    hipMemcpy(&hi, global_pos_acc.ptr(itr->hi), sizeof(Rect<1>), hipMemcpyDeviceToHost);
    total_nnzs += hi.hi[0] - lo.lo[0] + 1;
  }

  // Allocate our output buffers.
  auto pos_acc  = out_pos.create_output_buffer<Rect<1>, 1>(total_rows, true /* return_buffer */);
  auto crd_acc  = out_crd.create_output_buffer<coord_ty, 1>(total_nnzs, true /* return_buffer */);
  auto vals_acc = out_vals.create_output_buffer<val_ty, 1>(total_nnzs, true /* return_buffer */);

  // We'll start with a simple row-based parallelization for our copies. If/when performance
  // suffers due to this, we can think about algorithms for a full-data based parallelization.
  DeferredBuffer<size_t, 1> row_offsets({0, total_rows - 1}, Memory::GPU_FB_MEM);
  DeferredBuffer<Rect<1>, 1> rects({0, rects_cpu.size() - 1}, Memory::GPU_FB_MEM);
  hipMemcpy(
    rects.ptr(0), rects_cpu.data(), sizeof(Rect<1>) * rects_cpu.size(), hipMemcpyHostToDevice);
  auto blocks = get_num_blocks_1d(total_rows);
  calculate_copy_sizes<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
    total_rows, rects_cpu.size(), rects, row_offsets, global_pos_acc);
  // Scan over the counts to find the offsets for each row.
  ThrustAllocator alloc(Memory::GPU_FB_MEM);
  auto policy = thrust::cuda::par(alloc).on(stream);
  thrust::exclusive_scan(
    policy, row_offsets.ptr(0), row_offsets.ptr(0) + total_rows, row_offsets.ptr(0));
  // Perform the final scatter/gather.
  scatter_rows<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(total_rows,
                                                         rects_cpu.size(),
                                                         rects,
                                                         row_offsets,
                                                         global_pos_acc,
                                                         global_crd_acc,
                                                         global_vals_acc,
                                                         pos_acc,
                                                         crd_acc,
                                                         vals_acc);
  CHECK_CUDA_STREAM(stream);
}

}  // namespace sparse
